#include "hip/hip_runtime.h"
#include "stdafx.h" // Pre-compiled header.

#include <stdio.h>
#include <hip/hip_runtime.h>
#include "cudaforces.cuh"
//#include "constants.h"


// Kernel that executes on the CUDA device
__global__ void CoulombForce(float *ForceX, float *ForceY, float *ForceZ, const float *PosX,const float *PosY,const float *PosZ,const float *Charge, int N)
{

	// Constants
	 float fPI = acos(-1.0);
	 float feps0 = 8.854187817e-12; // Vacuum permittivity
	 float fe = 1.602176487e-19; // electron charge in C



	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	ForceX[idx] = 0.0;
	ForceY[idx] = 0.0;
	ForceZ[idx] = 0.0;
	// do the first "half"
	for(int n = 0; n  < idx; n++)
	{
		float dcubed = pow(pow(PosX[n]-PosX[idx], 2)+pow(PosY[n]-PosY[idx], 2)+pow(PosZ[n]-PosZ[idx], 2),((float) 1.5));
		ForceX[idx] += (PosX[idx]-PosX[n])/dcubed  *(fe*Charge[idx])*(fe*Charge[n])/(4*fPI*feps0);
		ForceY[idx] += (PosY[idx]-PosY[n])/dcubed  *(fe*Charge[idx])*(fe*Charge[n])/(4*fPI*feps0);
		ForceZ[idx] += (PosZ[idx]-PosZ[n])/dcubed  *(fe*Charge[idx])*(fe*Charge[n])/(4*fPI*feps0);

	}

	// do the rest
	for(int n = idx+1; n < N; n++)
	{
		float dcubed = pow(pow(PosX[n]-PosX[idx], 2)+pow(PosY[n]-PosY[idx], 2)+pow(PosZ[n]-PosZ[idx], 2),((float) 1.5));
		ForceX[idx] += (PosX[idx]-PosX[n])/dcubed  *(fe*Charge[idx])*(fe*Charge[n])/(4*fPI*feps0);
		ForceY[idx] += (PosY[idx]-PosY[n])/dcubed  *(fe*Charge[idx])*(fe*Charge[n])/(4*fPI*feps0);
		ForceZ[idx] += (PosZ[idx]-PosZ[n])/dcubed  *(fe*Charge[idx])*(fe*Charge[n])/(4*fPI*feps0);

	}

	// fixing the units
	//ForceX[idx] = ForceX[idx]*fe*fe/(4*fPI*feps0);
	//ForceY[idx] = ForceY[idx]*fe*fe/(4*fPI*feps0);
	//ForceZ[idx] = ForceZ[idx]*fe*fe/(4*fPI*feps0);
}

/*
void CoulombWrapper(float * ForceX, float * ForceY, float * ForceZ, const float * PosX,const float * PosY,const float * PosZ,const int Nions)
{
	//printf("Hello World\nThis is the CU file\n");


	float *ForceX_d, *ForceY_d, *ForceZ_d, *PosX_d, *PosY_d, *PosZ_d; // pointers to device
	size_t size = Nions * sizeof(float);
	hipMalloc((void **) &ForceX_d, size);   // Allocate array on device
	hipMalloc((void **) &ForceY_d, size);   // Allocate array on device
	hipMalloc((void **) &ForceZ_d, size);   // Allocate array on device
	hipMalloc((void **) &PosX_d, size);   // Allocate array on device
	hipMalloc((void **) &PosY_d, size);   // Allocate array on device
	hipMalloc((void **) &PosZ_d, size);   // Allocate array on device

	// copying data to device
	hipMemcpy(PosX_d, PosX, size, hipMemcpyHostToDevice);
	hipMemcpy(PosY_d, PosY, size, hipMemcpyHostToDevice);
	hipMemcpy(PosZ_d, PosZ, size, hipMemcpyHostToDevice);

	// Do calculation on device:
	int block_size = 4;
	int n_blocks = Nions/block_size + (Nions%block_size == 0 ? 0:1);
	CoulombForce <<< n_blocks, block_size >>> (ForceX_d, ForceY_d, ForceZ_d, PosX_d, PosY_d, PosZ_d, Nions);
	// Retrieve result from device and store it in host array
	hipMemcpy(ForceX, ForceX_d, sizeof(float)*Nions, hipMemcpyDeviceToHost);
	hipMemcpy(ForceY, ForceY_d, sizeof(float)*Nions, hipMemcpyDeviceToHost);
	hipMemcpy(ForceZ, ForceZ_d, sizeof(float)*Nions, hipMemcpyDeviceToHost);
	// Print results
	//for (int i=0; i<Nions; i++)
	//	  printf("%d %f\t %f\t %f\t\n", i, ForceX[i], ForceY[i], ForceZ[i]);
	// Cleanup
	hipFree(PosX_d);
	hipFree(PosY_d);
	hipFree(PosZ_d);
	hipFree(ForceX_d);
	hipFree(ForceY_d);
	hipFree(ForceZ_d);
}
*/
void FastCoulombWrapper(float * ForceX, float * ForceY, float * ForceZ,const float * PosX,const float * PosY,const float * PosZ, float * ForceX_d, float * ForceY_d, float * ForceZ_d, float * PosX_d, float * PosY_d, float * PosZ_d,const float * Charge,float * Charge_d, const int Nions)
{
	size_t size = Nions * sizeof(float);
	// copying data to device
	hipMemcpy(PosX_d, PosX, size, hipMemcpyHostToDevice);
	hipMemcpy(PosY_d, PosY, size, hipMemcpyHostToDevice);
	hipMemcpy(PosZ_d, PosZ, size, hipMemcpyHostToDevice);

	hipMemcpy(Charge_d, Charge, size, hipMemcpyHostToDevice);

	// Do calculation on device:
	int block_size = 32;
	int n_blocks = Nions/block_size + (Nions%block_size == 0 ? 0:1);
	CoulombForce <<< n_blocks, block_size >>> (ForceX_d, ForceY_d, ForceZ_d, PosX_d, PosY_d, PosZ_d,Charge_d, Nions);
	// Retrieve result from device and store it in host array
	hipMemcpy(ForceX, ForceX_d, sizeof(float)*Nions, hipMemcpyDeviceToHost);
	hipMemcpy(ForceY, ForceY_d, sizeof(float)*Nions, hipMemcpyDeviceToHost);
	hipMemcpy(ForceZ, ForceZ_d, sizeof(float)*Nions, hipMemcpyDeviceToHost);

	//for (int i=0; i<Nions; i++)
		//	  printf("%d %f\t %f\t %f\t\n", i, ForceX[i], ForceY[i], ForceZ[i]);

}

void CudaCoulombAlloc(float ** ForceX_d, float ** ForceY_d, float ** ForceZ_d, float ** PosX_d, float ** PosY_d, float ** PosZ_d, float ** Charge_d, const int Nions)
{


	size_t size = Nions * sizeof(float);
	hipMalloc((void **) ForceX_d, size);   // Allocate array on device
	hipMalloc((void **) ForceY_d, size);   // Allocate array on device
	hipMalloc((void **) ForceZ_d, size);   // Allocate array on device
	hipMalloc((void **) PosX_d, size);   // Allocate array on device
	hipMalloc((void **) PosY_d, size);   // Allocate array on device
	hipMalloc((void **) PosZ_d, size);   // Allocate array on device

	hipMalloc((void **) Charge_d, size);   // Allocate array on device
}

void CudaCoulombFree(float * ForceX_d, float * ForceY_d, float * ForceZ_d, float * PosX_d, float * PosY_d, float * PosZ_d, float * Charge_d)
{
	hipFree(PosX_d);
	hipFree(PosY_d);
	hipFree(PosZ_d);
	hipFree(ForceX_d);
	hipFree(ForceY_d);
	hipFree(ForceZ_d);

	hipFree(Charge_d);

	// Testing this magic clean up line...
	hipDeviceReset();
}



